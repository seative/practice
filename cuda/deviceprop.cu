#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "L2 Cache Size: " << deviceProp.l2CacheSize << " bytes" << std::endl;
        std::cout<< "device page-locked memory capabilities: "<<deviceProp.canMapHostMemory<<std::endl;
        std::cout << "The maximum amount of L2 cache that can be set-aside for persisting memory accesses: "<<deviceProp.persistingL2CacheMaxSize<< " bytes" << std::endl;
        std::cout << "The maximum size of the access policy window: "<<deviceProp.accessPolicyMaxWindowSize<< " bytes" << std::endl;
    
        std::cout<<"sharedMemPerBlock: "<<deviceProp.sharedMemPerBlock<< " bytes" << std::endl;
        std::cout<<"sharedMemPerMultiprocessor: "<<deviceProp.sharedMemPerMultiprocessor<< " bytes" << std::endl;
   
    }
    return 0;
}

