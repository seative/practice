#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <cassert>
#include <cmath>
#include <iostream>

struct Matrix {
  int width;
  int height;
  float* elements;
};

#define BLOCK_SIZE 16

__global__ void clear(Matrix C)
{
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  if(row<C.height && col<C.width)
  {
    C.elements[row*C.width+col]=0;
  }
}

__global__ void matMul_origin(Matrix A, Matrix B, Matrix C) {
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < A.height;
       row += blockDim.y * gridDim.y) {
    for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < B.width;
         col += blockDim.x * gridDim.x) {
      float Cvalue = 0;
      for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
      C.elements[row * C.width + col] = Cvalue;
    }
  }
}

__global__ void matMul_shared2(Matrix A, Matrix B, Matrix C) {
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  float Cvalue = 0;

  for (int m = 0; m < (A.width+BLOCK_SIZE-1) / BLOCK_SIZE; ++m) {
    int aRow = by * BLOCK_SIZE + ty;
    int aCol = m * BLOCK_SIZE + tx;
    int bRow = m * BLOCK_SIZE + ty;
    int bCol = bx * BLOCK_SIZE + tx;

    // if (aRow < A.height && aCol < A.width) {
    //   As[ty][tx] = A.elements[aRow * A.width + aCol];
    // } else {
    //   As[ty][tx] = 0.0f;
    // }
    As[ty][tx] = A.elements[aRow * A.width + aCol];

    // if (bRow < B.height && bCol < B.width) {
    //   Bs[tx][ty] = B.elements[bRow * B.width + bCol];
    // } else {
    //   Bs[tx][ty] = 0.0f;
    // }
    Bs[ty][tx] = B.elements[bRow * B.width + bCol];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Cvalue += As[ty][k] * Bs[k][tx];
    }
    __syncthreads();
  }
  // 将结果写入全局内存
  int cRow = by * BLOCK_SIZE + ty;
  int cCol = bx * BLOCK_SIZE + tx;
  // if (cRow < C.height && cCol < C.width) 
  {
    C.elements[cRow * C.width + cCol] = Cvalue;
  }
}

__global__ void matMul_shared(Matrix A, Matrix B, Matrix C) {
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  float Cvalue = 0;

  for (int m = 0; m < (A.width+BLOCK_SIZE-1) / BLOCK_SIZE; ++m) {
    int aRow = by * BLOCK_SIZE + ty;
    int aCol = m * BLOCK_SIZE + tx;
    int bRow = m * BLOCK_SIZE + tx;
    int bCol = bx * BLOCK_SIZE + ty;

    // if (aRow < A.height && aCol < A.width) {
    //   As[ty][tx] = A.elements[aRow * A.width + aCol];
    // } else {
    //   As[ty][tx] = 0.0f;
    // }
    As[ty][tx] = A.elements[aRow * A.width + aCol];

    // if (bRow < B.height && bCol < B.width) {
    //   Bs[tx][ty] = B.elements[bRow * B.width + bCol];
    // } else {
    //   Bs[tx][ty] = 0.0f;
    // }
    Bs[tx][ty] = B.elements[bRow * B.width + bCol];
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Cvalue += As[ty][k] * Bs[k][tx];
    }
    __syncthreads();
  }
  // 将结果写入全局内存
  int cRow = by * BLOCK_SIZE + ty;
  int cCol = bx * BLOCK_SIZE + tx;
  // if (cRow < C.height && cCol < C.width) 
  {
    C.elements[cRow * C.width + cCol] = Cvalue;
  }
}

const int BLOCK_SIZE_M=128;  // height of block of C that each  block calculate
const int BLOCK_SIZE_K =8 ;// width of block of A that each  block load into shared memory
const int BLOCK_SIZE_N=128;  // width of block of C that each  block calculate
const int THREAD_SIZE_Y=8; // height of block of C that each thread calculate
const int THREAD_SIZE_X=8;  // width of block of C that each thread calculate
// const bool ENABLE_DOUBLE_BUFFER=0; // whether enable double buffering or not


__global__ void matMul_register(Matrix A,Matrix B,Matrix C)
{
  __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
  __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];
  float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};

  int bx=blockIdx.x;
  int by=blockIdx.y;

  int tx=threadIdx.x;
  int ty=threadIdx.y;
  int tid=ty*blockDim.x+tx;

  int tNum=BLOCK_SIZE_M*BLOCK_SIZE_K/(blockDim.x*blockDim.y); 
  int stride= BLOCK_SIZE_M/tNum;

  int mapTx= tid%BLOCK_SIZE_K;
  int mapTy= tid/BLOCK_SIZE_K;

  for(int k=0;k< A.width/BLOCK_SIZE_K;++k)
  {
    int aRow= mapTy + by * BLOCK_SIZE_M;
    int aCol= mapTx + k * BLOCK_SIZE_K;
   
    int bRow= mapTx + k * BLOCK_SIZE_K;
    int bCol= mapTy + bx * BLOCK_SIZE_N;

    for(int i=0;i<tNum;++i)
    {
      As[mapTy+i*stride][mapTx]= A.elements[(aRow+i*stride) * A.width + aCol];
      Bs[mapTx][mapTy+i*stride]= B.elements[bRow * B.width + bCol + i*stride];
    }
    __syncthreads();

  // float regisM[THREAD_SIZE_Y][BLOCK_SIZE_K];
  // float regisN[BLOCK_SIZE_K][THREAD_SIZE_X];
  //   for(int j=0; j< THREAD_SIZE_Y; ++j)
  //   {
  //     for(int i=0; i< BLOCK_SIZE_K; ++i)
  //     {
  //        regisM[j][i]= As[ty*THREAD_SIZE_Y+j][i];
  //     }
  //   }

  //   for(int j=0; j< THREAD_SIZE_X; ++j)
  //   {
  //     for(int i=0; i< BLOCK_SIZE_K; ++i)
  //     {
  //        regisN[i][j]= Bs[i][tx*THREAD_SIZE_X+j];
  //     }
  //   }

  //   for(int j=0; j< THREAD_SIZE_Y; ++j)
  //   {
  //     for(int i=0; i< THREAD_SIZE_X; ++i)
  //     {
  //       for(int l=0; l< BLOCK_SIZE_K; ++l)
  //       {
  //         accum[j][i]+= regisM[j][l]*regisN[l][i];
  //       }
  //     }
  //   }

  float regisM[THREAD_SIZE_Y];
  float regisN[THREAD_SIZE_X];

  for(int k=0;k<BLOCK_SIZE_K;++k)
  {
    for(int i=0; i< THREAD_SIZE_Y; ++i)
    {
      regisM[i]= As[ty*THREAD_SIZE_Y+i][k];
    }

    for(int i=0; i< THREAD_SIZE_X; ++i)
    {
      regisN[i]= Bs[k][tx*THREAD_SIZE_X+i];
    }

    for(int i=0; i< THREAD_SIZE_Y; ++i)
    {
      for(int j=0; j< THREAD_SIZE_X; ++j)
      {
        accum[i][j]+= regisM[i]*regisN[j];
      }
    }
  }

    __syncthreads();
  }

  int cRowF= by * BLOCK_SIZE_M + ty*THREAD_SIZE_Y ;
  int cColF= bx * BLOCK_SIZE_N + tx*THREAD_SIZE_X ;

  for(int j=0; j< THREAD_SIZE_Y; ++j)
  {
    for(int i=0; i< THREAD_SIZE_X; ++i)
    {
        int cRow= cRowF+j;
        int cCol= cColF+i;
        if(cRow < C.height && cCol < C.width)
        {
          C.elements[cRow * C.width + cCol] = accum[j][i];
        }
    }
  }
}


void matMulCublas(hipblasHandle_t& handle, Matrix d_A,Matrix d_B, Matrix d_C,float alpha=1.0f,float beta=0.0f)
 {
    // 调用 cuBLAS 函数进行矩阵乘法
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                d_C.width, d_C.height, d_A.width,
                &alpha,
                d_B.elements, d_B.width,
                d_A.elements, d_A.width,
                &beta,
                d_C.elements, d_C.width);
}

void matMul(const Matrix d_A, const Matrix d_B, Matrix d_C) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(d_B.width / dimBlock.x, d_A.height / dimBlock.y);

  // 记录开始时间
  hipEventRecord(start);
  matMul_origin<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  // 等待事件完成
  hipEventSynchronize(stop);
  // 计算时间差
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  // 打印执行时间
  std::cout << "Origin Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  clear<<<dimGrid, dimBlock>>>(d_C);
  hipEventRecord(start);
  matMul_shared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Shared Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  clear<<<dimGrid, dimBlock>>>(d_C);
  hipEventRecord(start);
  matMul_shared2<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Shared2 Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  dimGrid.x=d_A.width/BLOCK_SIZE_M;
  dimGrid.y=d_B.width/BLOCK_SIZE_N;

  dimBlock.x=BLOCK_SIZE_M/THREAD_SIZE_X;
  dimBlock.y=BLOCK_SIZE_N/THREAD_SIZE_Y;
  clear<<<dimGrid, dimBlock>>>(d_C);
  hipEventRecord(start);
  matMul_register<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Register Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  clear<<<dimGrid, dimBlock>>>(d_C);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipEventRecord(start);
  matMulCublas(handle,d_A, d_B, d_C);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Cublas Kernel execution time: " << milliseconds << " ms"
            << std::endl;
  hipblasDestroy(handle);

  // 销毁 CUDA 事件
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipDeviceSynchronize();
}

int main() {
  int M = 2048;
  int K = 2048;
  int N = 2048;
  // 分配主机内存
  Matrix A, B, C;
  A.height = M;
  A.width = K;
  B.height = K;
  B.width = N;
  C.height = M;
  C.width = N;

  A.elements = (float*)malloc(M * K * sizeof(float));
  B.elements = (float*)malloc(K * N * sizeof(float));
  C.elements = (float*)malloc(M * N * sizeof(float));

  // 初始化矩阵 A 和 B
  for (int i = 0; i < M * K; ++i) {
    A.elements[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < K * N; ++i) {
    B.elements[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // 创建 CUDA 事件
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  Matrix d_A;
  d_A.width = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  matMul(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // check the result
  Matrix C_cpu;
  C_cpu.height = M;
  C_cpu.width = N;
  C_cpu.elements = (float*)malloc(M * N * sizeof(float));
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float sum = 0;
      for (int k = 0; k < K; ++k) {
        sum += A.elements[i * K + k] * B.elements[k * N + j];
      }
      C_cpu.elements[i * N + j] = sum;
      // std::cout<<i * N + j<<" "<<C.elements[i * N + j]<<" "<<C_cpu.elements[i * N + j]<<std::endl;
      assert(fabs(C.elements[i * N + j] - C_cpu.elements[i * N + j]) < 1e-2);
    }
  }

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  // 释放主机内存
  free(A.elements);
  free(B.elements);
  free(C.elements);
  free(C_cpu.elements);
}
