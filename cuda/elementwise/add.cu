
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CEIL(a, b) ((a + b - 1) / (b))

#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

int N=1<<20;
int block_size = 1024;
int grid_size  = CEIL(N, block_size);

// 函数定义
__global__ void elementwise_add(float* a, float* b, float *c, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
    // printf("1");
}

__global__ void elementwise_add_float4(float* a, float* b, float *c, int N) {
    int idx = (blockDim.x * blockIdx.x + threadIdx.x) * 4;

    if (idx < N) {
        float4 tmp_a = FLOAT4(a[idx]);
        float4 tmp_b = FLOAT4(b[idx]);
        float4 tmp_c;
        tmp_c.x = tmp_a.x + tmp_b.x;
        tmp_c.y = tmp_a.y + tmp_b.y;
        tmp_c.z = tmp_a.z + tmp_b.z;
        tmp_c.w = tmp_a.w + tmp_b.w;
        FLOAT4(c[idx]) = tmp_c;
    }
}

int main()
{
  // 创建 CUDA 事件
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  float* a, *b, *c;
  hipMalloc(&a, N*sizeof(float));
  hipMalloc(&b, N*sizeof(float));
  hipMalloc(&c, N*sizeof(float));


  hipEventRecord(start);
  elementwise_add<<<grid_size, block_size>>>(a, b, c, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Native Kernel execution time: %f ms\n", milliseconds);

  hipEventRecord(start);
  elementwise_add_float4<<<grid_size / 4, block_size>>>(a, b, c, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Float4 Kernel execution time: %f ms\n", milliseconds);
}