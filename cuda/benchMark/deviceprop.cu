#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCK_SIZE 256
__global__ void dummyKernel() {
    // 空内核函数
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "L2 Cache Size: " << deviceProp.l2CacheSize << " bytes" << std::endl;
        std::cout<<  "device page-locked memory capabilities: "<<deviceProp.canMapHostMemory<<std::endl;
        std::cout << "The maximum amount of L2 cache that can be set-aside for persisting memory accesses: "<<deviceProp.persistingL2CacheMaxSize<< " bytes" << std::endl;
        std::cout << "The maximum size of the access policy window: "<<deviceProp.accessPolicyMaxWindowSize<< " bytes" << std::endl;
    
        std::cout<<"sharedMemPerBlock: "<<deviceProp.sharedMemPerBlock<< " bytes" << std::endl;
        std::cout<<"sharedMemPerMultiprocessor: "<<deviceProp.sharedMemPerMultiprocessor<< " bytes" << std::endl;

         // 计算Occupancy
        int maxActiveBlocks;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, dummyKernel, BLOCK_SIZE, 0);
        int maxWarpsPerMultiprocessor = deviceProp.maxThreadsPerMultiProcessor / 32;
        std::cout << "Max active blocks: " << maxActiveBlocks << "  max warps per multiprocessor: " << maxWarpsPerMultiprocessor << std::endl;
        float occupancy = (maxActiveBlocks * (BLOCK_SIZE / 32)) / (float)maxWarpsPerMultiprocessor;
        std::cout << "Occupancy: " << occupancy * 100 << "%" << std::endl;

        // 查询内存带宽
        float memoryBandwidth = (deviceProp.memoryClockRate * 2 * (deviceProp.memoryBusWidth / 8)) / 1.0e6;
        std::cout << "Memory Bandwidth: " << memoryBandwidth << " GB/s" << std::endl;
   
    }
    return 0;
}

