#include <iostream>
#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void kernel(int* data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        data[idx] = idx;
    }
}

int main() {
    const int size = 1024;
    int* h_data = nullptr;
    int* d_data = nullptr;

    // 启用页锁定内存映射
    gpuErrChk(hipSetDeviceFlags(hipDeviceMapHost));

    // 分配页锁定内存，并映射到设备地址空间
    gpuErrChk(hipHostAlloc(&h_data, size * sizeof(int), hipHostMallocMapped));

    // 获取设备指针
    gpuErrChk(hipHostGetDevicePointer(&d_data, h_data, 0));

    // 初始化页锁定内存
    for (int i = 0; i < size; ++i) {
        h_data[i] = 0;
    }

    // 启动 CUDA 内核
    kernel<<<(size + 255) / 256, 256>>>(d_data, size);

    // 检查内核启动错误
    gpuErrChk(hipGetLastError());

    // 同步设备，确保内核执行完成
    gpuErrChk(hipDeviceSynchronize());

    // 输出结果
    for (int i = 0; i < size; ++i) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // 释放内存
    gpuErrChk(hipHostFree(h_data));

    return 0;
}
