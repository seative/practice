#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <iostream>

template <typename T>
struct CudaAllocator
{
    using value_type = T;

    T *allocate(size_t size)
    {
        T *ptr = nullptr;
        hipMallocManaged(&ptr, size * sizeof(T));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0)
    {
        hipFree(ptr);
    }

    template <typename... Args>
    void construct(T *p, Args &&...args)
    {
        if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
            ::new ((void *)p) T(std::forward<Args>(args)...);
    }
};

template <int N, class T>
__global__ void kernel(T *arr)
{
    // int i = blockDim.x * blockIdx.x + threadIdx.x;
    // if (i > n)
    //     return;
    // arr[i] = i;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        arr[i] = i;
    }
}

template <class Func>
__global__ void parallel_for(int n, Func func)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        func(i);
    }
}

struct MyFunctor
{
    __device__ void operator()(int i) const
    {
        printf("number %d\n", i);
    }
};

int main()
{
    int n = 1 << 25;
    float a = 3.14f;
    std::vector<float, CudaAllocator<float>> x(n);
    std::vector<float, CudaAllocator<float>> y(n);

    for (int i = 0; i < n; ++i)
    {
        x[i] = std::rand() * (1.f / RAND_MAX);
        y[i] = std::rand() * (1.f / RAND_MAX);
    }

    parallel_for<<<n / 512, 128>>>(n, [a, x = x.data(), y = y.data()] __device__(int i)
                                   { x[i] = a * x[i] + y[i]; });

    hipDeviceSynchronize();

    for (int i = 0; i < n; ++i)
    {
        printf("x[%d] =%f\n", i, x[i]);
    }
    // int n = 1 << 25;
    // std::vector<float, CudaAllocator<float>> arr(n);
    // std::vector<float> cpu(n);

    // std::chrono::time_point tick1 = std::chrono::system_clock::now();
    // // parallel_for<<<32, 128>>>(n, MyFunctor{});
    // for (int i = 0; i < n; ++i)
    //     cpu[i] = sinf(i);
    // std::chrono::time_point tock1 = std::chrono::system_clock::now();
    // std::cout << (tock1 - tick1).count() << std::endl;

    // std::chrono::system_clock::time_point tick2 = std::chrono::system_clock::now();
    // // parallel_for<<<32, 128>>>(n, MyFunctor{});
    // parallel_for<<<n / 512, 128>>>(n, [arr = arr.data()] __device__(int i)
    //                                { arr[i] = __sinf(i); });
    // cudaDeviceSynchronize();
    // std::chrono::system_clock::time_point tock2 = std::chrono::system_clock::now();
    // std::cout << (tock2 - tick2).count() << std::endl;
    // constexpr int n = 65535;
    // std::vector<int, CudaAllocator<int>> arr(n);
    // // int nthreads = 128;
    // // int nblocks = (n + nthreads - 1) / nthreads;
    // // int *arr;
    // // cudaMallocManaged(arr, n * sizeof(int));
    // kernel<n><<<32, 128>>>(arr.data());
    // cudaDeviceSynchronize();

    // for (int i = 0; i < n; ++i)
    // {
    //     printf("arr[%d]: %d\n", i, arr[i]);
    // }
    // // cudaFree(arr);
    return 0;
}