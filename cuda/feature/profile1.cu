#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void test1(float* C, int max)
{
 int threadId=blockIdx.x*blockDim.x+threadIdx.x;
 if(threadId<max)
 {
   C[threadId]=(float)(threadId);
 }
}

__global__ void test2(float* C, int max)
{
 int threadId=blockIdx.x*blockDim.x+threadIdx.x;
 if(threadId<max)
 {
   C[threadId*129]=(float)(threadId);
 }
}

__global__ void test3(float* C, int max)
{
 int threadId=blockIdx.x*blockDim.x+threadIdx.x;
 if(threadId<max)
 {
   C[threadId*7]=(float)(threadId);
 }
}


int main()
{
  thrust::device_vector<float>  d_c(16*256*129); 
  test1<<<16,256>>>(thrust::raw_pointer_cast(d_c.data()),16*256);
  test2<<<16,256>>>(thrust::raw_pointer_cast(d_c.data()),16*256);
  test3<<<16,256>>>(thrust::raw_pointer_cast(d_c.data()),16*256);
  hipDeviceSynchronize();
}