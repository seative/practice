#include <hip/hip_runtime.h>
#include <iostream>



__global__ void hist_kernel(int *bins, const int nbins, const int *__restrict__ input, size_t array_size)
{
    extern __shared__ int shared_bins[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=threadIdx.x; i<nbins; i+=blockDim.x)
    {
        shared_bins[i]=0;
    }
    __syncthreads();
    for (int i = tid; i < array_size; i += stride)
    {
        int bin = input[i];
        if (bin < 0 || bin >= nbins)
        {
            printf("bin %d out of range\n", bin);
        }
        atomicAdd(&shared_bins[bin], 1);
    }
    __syncthreads();

    for(int i=threadIdx.x;i<nbins;i+=blockDim.x)
    {
        atomicAdd(&bins[i],shared_bins[i]);
    }
}

int main()
{
    int arraysize=1<<8;
    int * input=(int *)malloc(arraysize*sizeof(int));

    for(int i=0;i<arraysize;i++)
    {
        input[i]=rand()%256;
    }
    
    int * bins;
    hipMallocManaged(&bins,256*sizeof(int));
    hist_kernel<<<1,256,256*sizeof(int)>>>(bins,256,input,arraysize);
}