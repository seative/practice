#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

template <typename T>
struct CudaAllocator
{
    using value_type = T;

    T *allocate(size_t size)
    {
        T *ptr = nullptr;
        hipMallocManaged(&ptr, size * sizeof(T));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0)
    {
        hipFree(ptr);
    }

    template <typename... Args>
    void construct(T *p, Args &&...args)
    {
        if constexpr (!(sizeof...(Args) == 0 && std::is_pod_v<T>))
            ::new ((void *)p) T(std::forward<Args>(args)...);
    }
};

__global__ void parallel_sum1(int *sum, int const *arr, int n)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        sum[0] += arr[i];
        // atomicAdd(&sum[0], arr[i]);
    }
}

__global__ void parallel_sum2(int *sum, int const *arr, int n)
{
    int local_sum = 0;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        local_sum += arr[i];
    }
    atomicAdd(&sum[0], local_sum);
}

__global__ void parallel_filter(int *sum, int *res, int const *arr, int n)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        if (arr[i] >= 2)
        {
            int loc = atomicAdd(&sum[0], 1);
            res[loc] = arr[i];
        }
    }
}

__device__ __inline__ int my_atomic_add(int *dst, int src)
{
    int old = *dst, expect;
    do
    {
        expect = old;
        old = atomicCAS(dst, expect, expect + src);
    } while (expect != old);
    return old;
}

__global__ void parallel_sum3(int *sum, int const *arr, int n)
{
    int local_sum = 0;
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        local_sum += arr[i];
    }
    my_atomic_add(&sum[0], local_sum);
}

__global__ void parallel_sum4(int *sum, int const *arr, int n)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n / 1024; i += blockDim.x * gridDim.x)
    {
        int local_sum = 0;
        for (int j = i * 1024; j < i * 1024 + 1024; j++)
        {
            local_sum += arr[j];
        }
        sum[i] = local_sum;
    }
}

__global__ void parallel_sum5(int *sum, int const *arr, int n)
{
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < n / 1024; i += blockDim.x * gridDim.x)
    {
        int local_sum[1024];
        for (int j = 0; j < 1024; j++)
        {
            local_sum[j] = arr[i * 1024 + j];
        }
        for (int j = 0; j < 512; j++)
        {
            local_sum[j] += local_sum[j + 512];
        }
        for (int j = 0; j < 256; j++)
        {
            local_sum[j] += local_sum[j + 256];
        }
        for (int j = 0; j < 128; j++)
        {
            local_sum[j] += local_sum[j + 128];
        }
        for (int j = 0; j < 64; j++)
        {
            local_sum[j] += local_sum[j + 64];
        }
        for (int j = 0; j < 32; j++)
        {
            local_sum[j] += local_sum[j + 32];
        }
        for (int j = 0; j < 16; j++)
        {
            local_sum[j] += local_sum[j + 16];
        }
        for (int j = 0; j < 8; j++)
        {
            local_sum[j] += local_sum[j + 8];
        }
        for (int j = 0; j < 4; j++)
        {
            local_sum[j] += local_sum[j + 4];
        }
        for (int j = 0; j < 2; j++)
        {
            local_sum[j] += local_sum[j + 2];
        }
        for (int j = 0; j < 1; j++)
        {
            local_sum[j] += local_sum[j + 1];
        }
        sum[i] = local_sum[0];
    }
}

__global__ void parallel_sum6(int *sum, int const *arr, int n)
{
    __shared__ volatile int local_sum[1024];
    int j = threadIdx.x;
    int i = blockIdx.x;
    int temp_sum = 0;

    for (int t = i * 1024 + j; t < n; t += 1024 * gridDim.x)
    {
        temp_sum += arr[t];
    }

    local_sum[j] = temp_sum;
    __syncthreads();
    if (j < 512)
    {
        local_sum[j] += local_sum[j + 512];
    }
    __syncthreads();
    if (j < 256)
    {
        local_sum[j] += local_sum[j + 256];
    }
    __syncthreads();
    if (j < 128)
    {
        local_sum[j] += local_sum[j + 128];
    }
    __syncthreads();
    if (j < 64)
    {
        local_sum[j] += local_sum[j + 64];
    }
    __syncthreads();
    if (j < 32)
    {
        local_sum[j] += local_sum[j + 32];
        local_sum[j] += local_sum[j + 16];
        local_sum[j] += local_sum[j + 8];
        local_sum[j] += local_sum[j + 4];
        local_sum[j] += local_sum[j + 2];
        if (j == 0)
        {
            sum[i] = local_sum[0] + local_sum[1];
        }
    }
}

int main()
{
    int n = 1 << 24;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> sum(n / 1024);

    for (int i = 0; i < n; ++i)
    {
        arr[i] = std::rand() % 4;
    }

    std::chrono::time_point tick1 = std::chrono::system_clock::now();
    // parallel_sum5<<<n / 1024, 1024>>>(sum.data(), arr.data(), n);
    parallel_sum6<<<n / 4096, 1024>>>(sum.data(), arr.data(), n);
    hipDeviceSynchronize();

    int final_sum = 0;
    for (int i = 0; i < n / 1024; ++i)
    {
        final_sum += sum[i];
    }
    std::chrono::time_point tick2 = std::chrono::system_clock::now();
    printf("result:%d\n", final_sum);
    std::cout << (double)std::chrono::duration_cast<std::chrono::microseconds>(tick2 - tick1).count() / 1e6 << " sec" << std::endl;
    return 0;
}