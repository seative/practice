#include <hip/hip_runtime.h>

#include <cassert>
#include <cmath>
#include <iostream>

struct Matrix {
  int width;
  int height;
  float* elements;
};

#define BLOCK_SIZE 16

__global__ void matMul_origin(Matrix A, Matrix B, Matrix C) {
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  for (int row = blockIdx.y * blockDim.y + threadIdx.y; row < A.height;
       row += blockDim.y * gridDim.y) {
    for (int col = blockIdx.x * blockDim.x + threadIdx.x; col < B.width;
         col += blockDim.x * gridDim.x) {
      float Cvalue = 0;
      for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
      C.elements[row * C.width + col] = Cvalue;
    }
  }
}

__global__ void matMul_shared(Matrix A, Matrix B, Matrix C) {
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  float Cvalue = 0;
  for (int m = 0; m < (A.width+BLOCK_SIZE-1) / BLOCK_SIZE; ++m) {
    int aRow = by * BLOCK_SIZE + ty;
    int aCol = m * BLOCK_SIZE + tx;
    int bRow = m * BLOCK_SIZE + ty;
    int bCol = bx * BLOCK_SIZE + tx;

    if (aRow < A.height && aCol < A.width) {
      As[ty][tx] = A.elements[aRow * A.width + aCol];
    } else {
      As[ty][tx] = 0.0f;
    }

    if (bRow < B.height && bCol < B.width) {
      Bs[ty][tx] = B.elements[bRow * B.width + bCol];
    } else {
      Bs[ty][tx] = 0.0f;
    }
    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Cvalue += As[ty][k] * Bs[k][tx];
    }
    __syncthreads();
  }
  // 将结果写入全局内存
  int cRow = by * BLOCK_SIZE + ty;
  int cCol = bx * BLOCK_SIZE + tx;
  if (cRow < C.height && cCol < C.width) {
    C.elements[cRow * C.width + cCol] = Cvalue;
  }
}

const int BLOCK_SIZE_M=128;  // height of block of C that each  block calculate
const int BLOCK_SIZE_K =8 ;// width of block of A that each  block load into shared memory
const int BLOCK_SIZE_N=128;  // width of block of C that each  block calculate
const int THREAD_SIZE_Y=8; // height of block of C that each thread calculate
const int THREAD_SIZE_X=8;  // width of block of C that each thread calculate
const bool ENABLE_DOUBLE_BUFFER=0; // whether enable double buffering or not


__global__ void matMul_register(Matrix A,Matrix B,Matrix C)
{
  __shared__ float As[BLOCK_SIZE_M][BLOCK_SIZE_K];
  __shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];

  int bx=blockIdx.x;
  int by=blockIdx.y;

  int tx=threadIdx.x;
  int ty=threadIdx.y;
  int tid=ty*blockDim.x+tx;
  int tNum=BLOCK_SIZE_M*BLOCK_SIZE_K/(blockDim.x*blockDim.y); 

  int stride= BLOCK_SIZE_M/tNum;

  // tNum/8

  float Cvalue = 0;
  for(int k=0;k< A.width/BLOCK_SIZE_K;++k)
  {
    int aCol= tid % BLOCK_SIZE_K+ k * BLOCK_SIZE_K;
    int aRow= tid / BLOCK_SIZE_K+ by * BLOCK_SIZE_M;

    int bRow= tid % BLOCK_SIZE_K+ k * BLOCK_SIZE_K;
    int bCol= tid / BLOCK_SIZE_K+ bx * BLOCK_SIZE_N;

    for(int i=0;i<tNum;++i)
    {
      As[tid/BLOCK_SIZE_K+i*stride][tid%BLOCK_SIZE_K]= A.elements[(aRow+i*stride) * A.width + aCol];
      Bs[tid%BLOCK_SIZE_K][tid/BLOCK_SIZE_K+i*stride]= B.elements[(bRow+i*stride) * B.width + bCol];
    }
    __syncthreads();
    for (int k = 0; k < BLOCK_SIZE; ++k) 
    {
      Cvalue += As[ty][k] * Bs[k][tx];
    }
    __syncthreads();
  }
}

void matMul(const Matrix d_A, const Matrix d_B, Matrix d_C) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(d_B.width / dimBlock.x, d_A.height / dimBlock.y);

  // 记录开始时间
  hipEventRecord(start);
  matMul_origin<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  // 等待事件完成
  hipEventSynchronize(stop);
  // 计算时间差
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  // 打印执行时间
  std::cout << "Origin Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  hipEventRecord(start);
  matMul_shared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << "Shared Kernel execution time: " << milliseconds << " ms"
            << std::endl;

  // 销毁 CUDA 事件
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

int main() {
  int M = 1024;
  int K = 1024;
  int N = 521;
  // 分配主机内存
  Matrix A, B, C;
  A.height = M;
  A.width = K;
  B.height = K;
  B.width = N;
  C.height = M;
  C.width = N;

  A.elements = (float*)malloc(M * K * sizeof(float));
  B.elements = (float*)malloc(K * N * sizeof(float));
  C.elements = (float*)malloc(M * N * sizeof(float));

  // 初始化矩阵 A 和 B
  for (int i = 0; i < M * K; ++i) {
    A.elements[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < K * N; ++i) {
    B.elements[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // 创建 CUDA 事件
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  Matrix d_A;
  d_A.width = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  matMul(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // check the result
  Matrix C_cpu;
  C_cpu.height = M;
  C_cpu.width = N;
  C_cpu.elements = (float*)malloc(M * N * sizeof(float));
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float sum = 0;
      for (int k = 0; k < K; ++k) {
        sum += A.elements[i * K + k] * B.elements[k * N + j];
      }
      C_cpu.elements[i * N + j] = sum;
      assert(fabs(C.elements[i * N + j] - C_cpu.elements[i * N + j]) < 1e-4);
    }
  }

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);

  // 释放主机内存
  free(A.elements);
  free(B.elements);
  free(C.elements);
  free(C_cpu.elements);
}
