#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <memory>

#include <iostream>
#include <nvfunctional>
#include <type_traits>

// 定义一个简单的类
// class MyClass {
// public:
//    __host__ __device__ MyClass(int value) : value(value) {}
//     __device__ void doubleValue() {
//         value *= 2;
//     }
//     __host__ __device__ int getValue() const {
//         return value;
//     }
// private:
//     int value;
// };

// CUDA 核函数，用于在设备内存上构造 MyClass 实例并处理
template <typename T>
__global__ void createAndProcessData(nvstd::function<void(int)>* obj, T func) {
  // 使用 placement new 在设备内存上构造 MyClass 实例
  new (obj) nvstd::function<void(int)>(func);
}

 template<typename Func, typename Ret, typename... Args>
    __global__ void create_device_funcWrapper(nvstd::function<Ret(Args...)>* wrapper, Func func)
    {
      // 使用 placement new 在设备内存上构造 nvstd::function 实例
      new (wrapper) nvstd::function<Ret(Args...)>(func);
    }

__global__ void test(nvstd::function<void(int)>* obj) {
  int i = threadIdx.x;
  obj->operator()(i);
}

template <typename T>
__global__ void test2(T func) {
  int i = threadIdx.x;
  func(i);
}

struct Cpu {};
struct Gpu {};

 template<class, class Fnsig>
  class FunctionWrapper
  {
    static_assert(std::is_same_v<Fnsig, Fnsig>, "not a valid function signature");
  };

  template<typename Ret, typename... Args>
  class FunctionWrapperBase
  {
  public:
    FunctionWrapperBase() = default;
    FunctionWrapperBase(const FunctionWrapperBase&) = delete;
    FunctionWrapperBase& operator=(const FunctionWrapperBase&) = delete;

    virtual ~FunctionWrapperBase() = default;
    virtual nvstd::function<Ret(Args...)>* get_function() const = 0;
  };

  template<typename Ret, typename... Args>
  class FunctionWrapper<Ret(Args...), Cpu> : public FunctionWrapperBase<Ret, Args...>
  {
  public:
    FunctionWrapper() = default;

    ~FunctionWrapper() override = default;

    template<typename Func>
    void wrap(Func func)
    {
      if constexpr (std::is_invocable_r_v<Ret, Func, Args...>)
      {
        if (hostFuncPtr)
          hostFuncPtr.reset();
        hostFuncPtr = std::make_unique<nvstd::function<Ret(Args...)>>(func);
      }
      else
      {
        static_assert(std::is_invocable_r_v<Ret, Func, Args...>, "Function signature does not match");
      }
    }

    nvstd::function<Ret(Args...)>* get_function() const override
    {
      return hostFuncPtr.get();
    }

  private:
    std::unique_ptr<nvstd::function<Ret(Args...)>> hostFuncPtr;
  };

  template<typename Ret, typename... Args>
  class FunctionWrapper<Ret(Args...), Gpu> : public FunctionWrapperBase<Ret, Args...>
  {
  public:
    FunctionWrapper() = default;

    ~FunctionWrapper() override
    {
      if (deviceFuncPtr)
      {
        hipFree(deviceFuncPtr);
      }
    }

    template<typename Func>
    void wrap(Func func)
    {
      if constexpr (std::is_invocable_r_v<Ret, Func, Args...>)
      {
        if (deviceFuncPtr)
        {
          hipFree(deviceFuncPtr);
          deviceFuncPtr = nullptr;
        }
        hipError_t err = hipMalloc(&deviceFuncPtr, sizeof(nvstd::function<Ret(Args...)>));
        if (err != hipSuccess)
        {
          throw std::runtime_error("Failed to allocate device memory");
        }
        create_device_funcWrapper<<<1, 1>>>(deviceFuncPtr, func);
        err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
          hipFree(deviceFuncPtr);
          deviceFuncPtr = nullptr;
          throw std::runtime_error("Failed to synchronize device");
        }
      }
      else
      {
        static_assert(std::is_invocable_r_v<Ret, Func, Args...>, "Function signature does not match");
      }
    }

    nvstd::function<Ret(Args...)>* get_function() const override
    {
      return deviceFuncPtr;
    }

  private:
    nvstd::function<Ret(Args...)>* deviceFuncPtr = nullptr;
  };

  __device__ void deviceFunc(int x)
{
  printf("deviceFunc: %d\n", x*2);
}

// int main() {
//   // // 在主机上定义 MyClass 实例
//   // // nvstd::function<void(int)> h_obj
//   // nvstd::function<void(int)>* d_obj;

//   // // 在设备内存上分配 MyClass 实例的空间
//   // hipMalloc(&d_obj, sizeof(nvstd::function<void(int)>));
//   int a = 5;

//   // // 启动 CUDA 核函数，在设备内存上构造 MyClass 实例并处理
//   // createAndProcessData<<<1, 1>>>(d_obj, [a] __device__(int x) {
//   //   x = a + pow(x * 2, 1.3);
//   //   // printf("Function 2: %d\n", x);
//   // });

//   FunctionWrapper<void(int),Gpu> wrapper;
//   // wrapper.wrap([a] __device__(int x) {
//   //   printf("x: %d\n", x);
//   //   x = a + pow(x * 2, 1.3);
//   //   printf("Function 2: %d\n", x);
//   // });
//   // wrapper.wrap(deviceFunc);

//   FunctionWrapper<void(int),Cpu> wrapperhost;
//   wrapperhost.wrap([a](int x) {
//     x = a + pow(x * 2, 1.3);
//     printf("Function 2 host: %d\n", a);
//   });

//   // wrapperhost.getHostFunction()->operator()(5);

//   // 创建 CUDA 事件
//   float milliseconds = 0;
//   hipEvent_t start, end;
//   hipEventCreate(&start);
//   hipEventCreate(&end);

//   hipEventRecord(start);
//   test<<<1, 32>>>(wrapper.get_function());
//   hipEventRecord(end);
//   hipEventSynchronize(end);
//   hipEventElapsedTime(&milliseconds, start, end);
//   std::cout << "Kernel 1 execution time: " << milliseconds << " ms"
//             << std::endl;

//   // 同步设备
//   hipDeviceSynchronize();

//   hipEventRecord(start);
//   test2<<<1, 32>>>([a] __device__(int x) { x = a + pow(x * 2, 1.3); });
//   hipEventRecord(end);
//   hipEventSynchronize(end);

//   hipEventElapsedTime(&milliseconds, start, end);
//   std::cout << "Kernel 2 execution time: " << milliseconds << " ms"
//             << std::endl;
//   // printf("Function 2: %d\n", x);

//   // // 将结果从设备复制回主机
//   // hipMemcpy(&h_obj, d_obj, sizeof(MyClass), hipMemcpyDeviceToHost);

//   // // 输出结果
//   // std::cout << "Processed value: " << h_obj.getValue() << std::endl;

//   // 释放设备内存上的 MyClass 实例
//   // hipFree(d_obj);

//   return 0;
// }

// __global__ void test3(T func) {
//   int i = threadIdx.x;
//   func(i);
// }


template <typename T>
class base1 {
 protected:
  T b=10;

 public:
  int add(int a) { return a + b; }
};

template<typename T>
class son1 : public base1<T> {
  T x=0;
 public:
  int minus(int a) { return a - this->b; }
};


int main()
{
  son1<int> s;
  std::cout<<s.add(1);

}